// alloc.cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstddef>

extern "C" {
  // Allocate “size” bytes of host memory, mapped into the device’s address space
  void* my_alloc(ssize_t size, int device, hipStream_t stream) {
    void* ptr = nullptr;
    // Make sure we’re on the right GPU
    hipSetDevice(device);
    // Allocate pinned host memory that’s MAPPED into the CUDA UVA space
    hipHostAlloc(&ptr, size,
                  hipHostMallocMapped    // maps it into device VA space
                | hipHostMallocPortable // visible to all contexts
                );
    return ptr;
  }

  // Free it when PyTorch asks
  void my_free(void* ptr, size_t size, hipStream_t stream) {
    hipHostFree(ptr);
  }
}